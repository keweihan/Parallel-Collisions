#include "hip/hip_runtime.h"
#include "CudaResolve.h"
#include "ColliderCell.h"
#include "Core/Entity.h"
#include "Physics/PhysicsBody.h"
#include "Collision/BoxCollider.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace SimpleECS;

ColliderEntity::ColliderEntity(SimpleECS::BoxCollider* col) {
    Collider::AABB bounds;
    col->getBounds(bounds);

    x_max = bounds.xMax;
    x_min = bounds.xMin;
    y_min = bounds.yMin;
    y_max = bounds.yMin;
    
    y_pos = col->entity->transform->position.y;
    x_pos = col->entity->transform->position.x;

    y_vel = col->entity->getComponent<PhysicsBody>()->velocity.y;
    x_vel = col->entity->getComponent<PhysicsBody>()->velocity.x;
}

// Flatten and copy to device
void CudaResolve::flattenCopyToDevice() {
    std::vector<ColliderEntity> flattenedData;
    std::vector<int> lengths; // sizes of cells in flattened
    std::vector<int> offsets; // starts of cells in flattened
    
    int offset = 0;
    for (ColliderCell& cell : *_cells) {
        lengths.push_back(cell.size());
        offsets.push_back(offset);

        for(Collider* col : cell) {
            BoxCollider* box = static_cast<BoxCollider*>(col);
            ColliderEntity flatCollider(box);
            flattenedData.push_back(flatCollider);
        }
    }

    allocateAndCopyToDevice(flattenedData, lengths, offsets);
}

// Allocate and copy to device
void CudaResolve::allocateAndCopyToDevice(const std::vector<ColliderEntity>& flattenedData, 
                                          const std::vector<int>& lengths, 
                                          const std::vector<int>& offsets) {
    hipMalloc(&d_flattenedData, flattenedData.size() * sizeof(ColliderEntity));
    hipMalloc(&d_lengths, lengths.size() * sizeof(int));
    hipMalloc(&d_offsets, offsets.size() * sizeof(int));

    hipMemcpy(d_flattenedData, flattenedData.data(), flattenedData.size() * sizeof(ColliderEntity), hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths.data(), lengths.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice);
}

// Kernel
__global__ void kernel(ColliderEntity* d_flattenedData, int* d_lengths, int* d_offsets, int numCells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numCells) {
        int length = d_lengths[idx];
        int offset = d_offsets[idx];

        for (int i = 0; i < length; ++i) {
            ColliderEntity entity = d_flattenedData[offset + i];
            // Perform operations on entity
        }
    }
}

// Kernel launch
void CudaResolve::launchKernel(int numThreads) {
    int numCells = _cells->size();
    dim3 blockDim(256);
    dim3 gridDim((numCells + blockDim.x - 1) / blockDim.x);
    kernel<<<gridDim, blockDim>>>(d_flattenedData, d_lengths, d_offsets, numCells);
    hipDeviceSynchronize();  // Ensure kernel completion
}

// Destructor
CudaResolve::~CudaResolve() {
    if (d_flattenedData) hipFree(d_flattenedData);
    if (d_lengths) hipFree(d_lengths);
    if (d_offsets) hipFree(d_offsets);
}

